#include <hip/hip_runtime.h>
#include <stdio.h>
#include <random>
#include <iostream>

#include "matmul.cuh"

int main(int argc, char* argv[]){

    std::random_device entropy_source;
    std::mt19937 generator(entropy_source());
    std::uniform_real_distribution<float> distA(-1, 1);
    std::uniform_real_distribution<float> distB(-1, 1);

    // CUDA timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    std::size_t n = std::stoi(argv[1]); // get the number of elments of the array from the command line
    std::size_t threads_per_block = std::stoi(argv[2]); // get the number of threads per block from the command line

    // generate the random arrays a and b
    float *a = (float*) malloc(n * n * sizeof(float));
    float *b = (float*) malloc(n * n * sizeof(float));
    float *c = (float*) malloc(n * n * sizeof(float));

    float *dA, *dB, *dC;

    // allocate memory on the device
    hipMalloc((void**)&dA, sizeof(float) * n * n);
    hipMalloc((void**)&dB, sizeof(float) * n * n);
    hipMalloc((void**)&dC, sizeof(float) * n * n);

    // fill the arrays with random numbers corresponding to their range
    for(std::size_t i = 0; i < n * n; i++) {
        a[i] = distA(generator);
        b[i] = distB(generator);
    }

    // copy the randomly generated arrays to the device
    hipMemcpy(dA, a, n*n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, b, n*n*sizeof(float), hipMemcpyHostToDevice);
    hipMemset(dC, 0, n*n*sizeof(float));

    hipEventRecord(start);
    matmul(dA, dB, dC, n*n, threads_per_block);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // copy the results back to the host
    hipMemcpy(c, dC, n*n*sizeof(float), hipMemcpyDeviceToHost);
    hipEventElapsedTime(&elapsedTime, start, stop);

    std::cout << n << "\n" << std::endl;
    std::cout << c[(n*n)-1] << "\n" << std::endl;
    std::cout << elapsedTime << "\n" << std::endl;

    // clean up
    free(a);
    free(b);
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);

    // destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
