#include <random>
#include <cstdio>
#include <iostream>
#include <hip/hip_runtime.h>

#include "vscale.cuh"

const int THREADS = 512;

int main(int argc, char *argv[]) {
    // create generators for random numbers
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source());
    std::uniform_real_distribution<float> distA(-10, 10);
    std::uniform_real_distribution<float> distB(0, 1);
    
    // CUDA timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    std::size_t n = std::stoi(argv[1]); // get the number of elments from the command line

    // generate the random arrays a and b
    float *a = (float*) malloc(n * sizeof(float));
    float *b = (float*) malloc(n * sizeof(float));

    float *dA, *dB;

    // allocate memory on the device
    hipMalloc((void**)&dA, sizeof(float) * n);
    hipMalloc((void**)&dB, sizeof(float) * n);

    // fill the arrays with random numbers corresponding to their range
    for(std::size_t i = 0; i < n; i++) {
        a[i] = distA(generator);
        b[i] = distB(generator);
    }

    // copy the randomly generated arrays to the device
    hipMemcpy(dA, a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, b, n*sizeof(float), hipMemcpyHostToDevice);
    
    const int numberOfBlocks = (n + THREADS -1)/THREADS;

    hipEventRecord(start);
    vscale<<<numberOfBlocks,THREADS>>>(dA,dB,n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // copy the results back to the host
    hipMemcpy(b, dB, n*sizeof(float), hipMemcpyDeviceToHost); 
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    std::cout << elapsedTime << std::endl;
    std::cout << b[0] << std::endl;
    std::cout << b[n-1] << std::endl;
    std::cout << "\n";
    // clean up

    // destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // free memory
    hipFree(dA);
    hipFree(dB);
    free(a);
    free(b);

    return 0;
}
