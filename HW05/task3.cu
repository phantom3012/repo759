#include <random>
#include <stdio.h>

#include "vscale.cuh"

const int THREADS = 512;

int main(int argc, char *argv[]) {
    //create generators for random numbers
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source());
    std::uniform_real_distribution<float> distA(-10, 10);
    std::uniform_real_distribution<float> distB(0, 1);
    
    //CUDA timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    std::size_t n = std::stoi(argv[1]); //get the number of elments from the command line

    //generate the random arrays a and b
    float a[n];
    float b[n];

    //fill the arrays with random numbers corresponding to their range
    for(std::size_t i = 0; i < n; i++) {
        a[i] = distA(generator);
        b[i] = distB(generator);
    }

    const int numberOfBlocks = (n + THREADS -1)/THREADS;

    hipEventRecord(start);
    vscale<<<numberOfBlocks,THREADS>>>(a,b,n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);
    
    printf("%lu\n", n);
    printf("%f\n", elapsedTime);
    printf("%f\n", b[0]);
    printf("%f\n", b[n-1]); 

    //clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}
