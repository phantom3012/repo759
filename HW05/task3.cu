#include <iostream>
#include <random>

#include "vscale.cu"

const int THREADS = 512;

int main(int argc, char *argv[]) {
    //create generators for random numbers
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source());
    std::uniform_real_distribution<float> distA(-10, 10);
    std::uniform_real_distribution<float> distB(-1, 1);

    //CUDA timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    int n = std::stoi(argv[1]); //get the number of elments from the command line

    //generate the random arrays a and b
    const float *a = (float*) malloc(n*sizeof(float)) ;
    float *b = (float*) malloc(n*sizeof(float));

    //fill the arrays with random numbers corresponding to their range
    for(int i = 0; i < n; i++) {
        a[i] = distA(generator);
        b[i] = distB(generator);
    }

    for(int i = 0; i < n; i++) {
	    std::cout << "a[" << i << "] = " << a[i] << "\n";
	    std::cout << "b[" << i << "] = " << b[i] << "\n";
    }

    int numberOfBlocks = (n + THREADS -1)/THREADS;

    hipEventRecord(start);
    vscale<<<numberOfBlocks,THREADS>>>(a,b,n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    hipEventElapsedTime(&elapsedTime, start, stop);

    std:: cout << elapsedTime << "\n";
    std::cout << b[0] << "\n";
    std::cout << b[n-1] << "\n";

    //clean up
    hipEventDestroy(start);
    hipEventDestroy(stop);
    free(a);
    free(b);

    return 0;
}
