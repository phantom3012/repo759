#include <random>
#include <stdio.h>

#include "vscale.cuh"

const int THREADS = 512;

int main(int argc, char *argv[]) {
    // create generators for random numbers
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source());
    std::uniform_real_distribution<float> distA(-10, 10);
    std::uniform_real_distribution<float> distB(0, 1);
    
    // CUDA timing events
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float elapsedTime;

    std::size_t n = std::stoi(argv[1]); // get the number of elments from the command line

    // generate the random arrays a and b
    float *a = malloc(n * sizeof(float));
    float *b = malloc(n * sizeof(float));

    float *dA, *dB;

    // allocate memory on the device
    hipMalloc((void**)&dA, sizeof(float) * n);
    hipMalloc((void**)&dB, sizeof(float) * n);

    // fill the arrays with random numbers corresponding to their range
    for(std::size_t i = 0; i < n; i++) {
        a[i] = distA(generator);
        b[i] = distB(generator);
    }

    // copy the randomly generated arrays to the device
    hipMemcpy(dA, a, n*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dB, b, n*sizeof(float), hipMemcpyHostToDevice);
    
    const int numberOfBlocks = (n + THREADS -1)/THREADS;

    hipEventRecord(start);
    vscale<<<numberOfBlocks,THREADS>>>(dA,dB,n);
    hipEventRecord(stop);
    hipEventSynchronize(stop);

    // copy the results back to the host
    hipMemcpy(b, dB, n*sizeof(float), hipMemcpyDeviceToHost); 
    hipEventElapsedTime(&elapsedTime, start, stop);
    
    printf("%lu\n", n);
    printf("%f\n", elapsedTime);
    printf("%f\n", b[0]);
    printf("%f\n", b[n-1]); 

    // clean up

    // destroy events
    hipEventDestroy(start);
    hipEventDestroy(stop);

    // free memory
    hipFree(dA);
    hipFree(dB);
    free(a);
    free(b);

    return 0;
}
