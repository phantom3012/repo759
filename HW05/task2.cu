#include <hip/hip_runtime.h>
#include <iostream>
#include <random>

const int NUM_ELEMENTS = 16;
const int BLOCKS = 2;
const int THREADS = 8;

__global__ void weightedAddition(int a, int *dA) {
    int weightedSum = a * threadIdx.x + blockIdx.x;
    dA[THREADS*blockIdx.x + threadIdx.x] = weightedSum;
}

int main() {
    std::random_device entropy_source;
    std::mt19937 generator(entropy_source());
    std::uniform_int_distribution<int> dist(0, 10);

    int a = dist(generator); //generate the random number a

    int hA[NUM_ELEMENTS]; //host array
    int *dA; //device array

    hipMalloc((void**)&dA, sizeof(int)*NUM_ELEMENTS); //assign an int array of 16 on the device
    hipMemset(dA, 0, NUM_ELEMENTS*sizeof(int)); //set the device array to 0
    
    weightedAddition<<<BLOCKS,THREADS>>>(a,dA); //call the kernel function

    hipMemcpy(&hA, dA, sizeof(int)*NUM_ELEMENTS, hipMemcpyDeviceToHost); //copy device array to host array

    //print the host array
    for(int i = 0; i < NUM_ELEMENTS; i++) {
        std::cout << hA[i] << " ";
    }
    std::cout << "\n";
    hipFree(dA); //free the device array

    return 0;
}
