#include "hip/hip_runtime.h"
#include "vscale.cuh"

__global__ void vscale(const float *a, float *b, unsigned int n) {
    std::size_t i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) {
        b[i] *= a[i];
    }
}
